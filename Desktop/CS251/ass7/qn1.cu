
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define USAGE_EXIT(s) do{\
      printf("Usage: %s <#of elements> <# of rows> <# of cols> \n %s \n",argv[0],s);\
      exit(-1);\
      }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    double num1;
                    double num2;
                    double result;
};

//Possibly log n steps

__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}
__global__ void calculate(char *mem, int num)
{   //TODO: maybe to edit this block.
    //  int i = blockDim.x * blockIdx.x + threadIdx.x;//thread id 
      int i = blockDim.x*blockDim.y*blockIdx.x+blockDim.x*threadIdx.y + threadIdx.x; //XXX
      if(i >= num)
           return;
       struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
      function(a);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;
    int row,col;
    if ( argc !=4)
    {
        USAGE_EXIT("Invalid No. of Argument");
    }  
    else 
    {
        num = atoi(argv[1]);
        row = atoi(argv[2]);
        col = atoi(argv[3]);
        if(row <=0 ||col <=0||row*col > 1024)
          USAGE_EXIT("Row/Col number index out of range");
        if(num < 1)
          USAGE_EXIT("Invalid no. of  elements specified.");
    }
    //arguements are fine
    /* Allocate host (CPU) memory and initialize*/
    /* MAX THREADS =  row* col*/
    int gridthread=row*col; //XXX
    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr; 
    for(i=0; i<num; ++i){
       pa = (struct num_array *) sptr;
       pa->num1 = (double) i + (double) i * 0.1;
       pa->num2 = pa->num1 + 1.0;
       sptr += 3 * sizeof(double);
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num /gridthread;
    
    if(num % gridthread)
           ++blocks;
    /*Kernel Invocation*/
    dim3 nThreadsPerBlock(row,col);//XXX
      calculate<<<blocks, nThreadsPerBlock>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;
   
    /*Print the last element for sanity check*/ 
    pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);

    
    free(ptr);
}
